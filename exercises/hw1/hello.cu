
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x,threadIdx.x);
}

int main(){

  hello<<<2,2>>>();
  hipDeviceSynchronize();  //直到当前设备上所有的异步操作（包括核函数执行、内存传输等）都完成后，主机才会继续执行后续代码。
}

